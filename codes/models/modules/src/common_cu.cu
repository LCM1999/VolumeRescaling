#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
#include "common_h.h"

int calculateBlockNumber(unsigned long totalSize, int blockSize)
{
  int numberOfBlocks = totalSize/blockSize;
  if (totalSize % blockSize != 0)
  {
    ++numberOfBlocks;
  }
  return numberOfBlocks;
}

int calculateBlockNumber(int totalSize, int blockSize)
{
  return calculateBlockNumber((unsigned long) totalSize, blockSize);
}


