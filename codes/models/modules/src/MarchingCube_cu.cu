#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _MARCHING_CUBES_KERNEL_CU_
#define _MARCHING_CUBES_KERNEL_CU_

#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime_api.h"    // includes for helper CUDA functions
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

#include "defines.h"
#include "tables.h"

// textures containing look-up tables
texture<uint, 1, hipReadModeElementType> edgeTex;
texture<uint, 1, hipReadModeElementType> triTex;
texture<uint, 1, hipReadModeElementType> numVertsTex;

// volume data
texture<float, 1, hipReadModeElementType> volumeTex;

extern "C"
void allocateTextures(uint **d_edgeTable, uint **d_triTable,  uint **d_numVertsTable)
{
    checkCudaErrors(hipMalloc((void **) d_edgeTable, 256*sizeof(uint)));
    checkCudaErrors(hipMemcpy((void *)*d_edgeTable, (void *)edgeTable, 256*sizeof(uint), hipMemcpyHostToDevice));
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipBindTexture(0, edgeTex, *d_edgeTable, channelDesc));

    checkCudaErrors(hipMalloc((void **) d_triTable, 256*16*sizeof(uint)));
    checkCudaErrors(hipMemcpy((void *)*d_triTable, (void *)triTable, 256*16*sizeof(uint), hipMemcpyHostToDevice));
    checkCudaErrors(hipBindTexture(0, triTex, *d_triTable, channelDesc));

    checkCudaErrors(hipMalloc((void **) d_numVertsTable, 256*sizeof(uint)));
    checkCudaErrors(hipMemcpy((void *)*d_numVertsTable, (void *)numVertsTable, 256*sizeof(uint), hipMemcpyHostToDevice));
    checkCudaErrors(hipBindTexture(0, numVertsTex, *d_numVertsTable, channelDesc));
}

extern "C"
void bindVolumeTexture(float *d_volume)
{
    // bind to linear texture
    checkCudaErrors(hipBindTexture(0, volumeTex, d_volume, hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat)));
}

// sample volume data set at a point
__device__
float sampleVolume(uint3 p, uint3 gridSize)
{
    p.x = min(p.x, gridSize.x - 1);
    p.y = min(p.y, gridSize.y - 1);
    p.z = min(p.z, gridSize.z - 1);
    uint i = (p.z*gridSize.x*gridSize.y) + (p.y*gridSize.x) + p.x;
    //    return (float) data[i] / 255.0f;
    return tex1Dfetch(volumeTex, i);
}

// compute position in 3d grid from 1d index
// only works for power of 2 sizes
__device__
uint3 calcGridPos(uint i, uint3 gridSize)
{
    uint3 gridPos;
    gridPos.x = i % gridSize.x;
    gridPos.y = (i / gridSize.x) % gridSize.y;
    gridPos.z = i/(gridSize.x*gridSize.y);
    return gridPos;
}

// classify voxel based on number of vertices it will generate
// one thread per voxel
__global__ void
classifyVoxel(uint *voxelVerts, uint *voxelOccupied, float *volume, 
			  uint3 gridSize, float isoValue)
{
	const uint numVoxels = gridSize.x*gridSize.y*gridSize.z;

    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

	if (i>=numVoxels) return;

    uint3 gridPos = calcGridPos(i, gridSize);

    // read field values at neighbouring grid vertices
    float field[8];
    field[0] = sampleVolume(gridPos, gridSize);
    field[1] = sampleVolume(gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(gridPos + make_uint3(0, 1, 1), gridSize);

    // calculate flag indicating if each vertex is inside or outside isosurface
    uint cubeindex;
    cubeindex =  uint(field[0] < isoValue);
    cubeindex += uint(field[1] < isoValue)*2;
    cubeindex += uint(field[2] < isoValue)*4;
    cubeindex += uint(field[3] < isoValue)*8;
    cubeindex += uint(field[4] < isoValue)*16;
    cubeindex += uint(field[5] < isoValue)*32;
    cubeindex += uint(field[6] < isoValue)*64;
    cubeindex += uint(field[7] < isoValue)*128;

    // read number of vertices from texture
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    voxelVerts[i] = numVerts;
	voxelOccupied[i] = (numVerts > 0);
}

extern "C" void
launch_classifyVoxel(dim3 grid, dim3 threads, uint *voxelVerts, uint *voxelOccupied, float *volume,
                     uint3 gridSize, float isoValue)
{
    // calculate number of vertices need per voxel
    classifyVoxel<<<grid, threads>>>(voxelVerts, voxelOccupied, volume,
                                     gridSize, isoValue);
    getLastCudaError("classifyVoxel failed");
}


// compact voxel array
__global__ void
compactVoxels(uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (voxelOccupied[i] && (i < numVoxels))
    {
        compactedVoxelArray[ voxelOccupiedScan[i] ] = i;
    }
}

extern "C" void
launch_compactVoxels(dim3 grid, dim3 threads, uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    compactVoxels<<<grid, threads>>>(compactedVoxelArray, voxelOccupied,
                                     voxelOccupiedScan, numVoxels);
    getLastCudaError("compactVoxels failed");
}

// compute interpolated vertex along an edge
__device__
float3 vertexInterp(float isolevel, float3 p0, float3 p1, float f0, float f1)
{
    float t = (isolevel - f0) / (f1 - f0);
    return lerp(p0, p1, t);
}

// compute interpolated vertex position and normal along an edge
__device__
void vertexInterp2(float isolevel, float3 p0, float3 p1, float4 f0, float4 f1, float3 &p, float3 &n)
{
    float t = (isolevel - f0.w) / (f1.w - f0.w);
    p = lerp(p0, p1, t);
    n.x = lerp(f0.x, f1.x, t);
    n.y = lerp(f0.y, f1.y, t);
    n.z = lerp(f0.z, f1.z, t);
    //    n = normalize(n);
}

// calculate triangle normal
__device__
float3 calcNormal(float3 *v0, float3 *v1, float3 *v2)
{
    float3 edge0 = *v1 - *v0;
    float3 edge1 = *v2 - *v0;
    // note - it's faster to perform normalization in vertex shader rather than here
    return cross(edge0, edge1);
}

// version that calculates flat surface normal for each triangle
__global__ void
generateTriangles(float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, uchar *volume,
                   uint3 gridSize, float isoValue, uint activeVoxels, uint maxVerts)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (i > activeVoxels - 1)
    {
        i = activeVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint voxel = compactedVoxelArray[i];
#else
    uint voxel = i;
#endif

    // compute position in 3d grid
    uint3 gridPos = calcGridPos(voxel, gridSize);

    float3 p;
    p.x = gridPos.x;
    p.y = gridPos.y;
    p.z = gridPos.z;

    // calculate cell vertex positions
    float3 v[8];
    v[0] = p;
    v[1] = p + make_float3(1.0f, 0, 0);
    v[2] = p + make_float3(1.0f, 1.0f, 0);
    v[3] = p + make_float3(0, 1.0f, 0);
    v[4] = p + make_float3(0, 0, 1.0f);
    v[5] = p + make_float3(1.0f, 0, 1.0f);
    v[6] = p + make_float3(1.0f, 1.0f, 1.0f);
    v[7] = p + make_float3(0, 1.0f, 1.0f);

    float field[8];
    field[0] = sampleVolume(gridPos, gridSize);
    field[1] = sampleVolume(gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(gridPos + make_uint3(0, 1, 1), gridSize);

    // recalculate flag
    uint cubeindex;
    cubeindex =  uint(field[0] < isoValue);
    cubeindex += uint(field[1] < isoValue)*2;
    cubeindex += uint(field[2] < isoValue)*4;
    cubeindex += uint(field[3] < isoValue)*8;
    cubeindex += uint(field[4] < isoValue)*16;
    cubeindex += uint(field[5] < isoValue)*32;
    cubeindex += uint(field[6] < isoValue)*64;
    cubeindex += uint(field[7] < isoValue)*128;

    // find the vertices where the surface intersects the cube

#if USE_SHARED
    // use shared memory to avoid using local
    __shared__ float3 vertlist[12*NTHREADS];

    vertlist[threadIdx.x] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[NTHREADS+threadIdx.x] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[(NTHREADS*2)+threadIdx.x] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[(NTHREADS*3)+threadIdx.x] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);
    vertlist[(NTHREADS*4)+threadIdx.x] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[(NTHREADS*5)+threadIdx.x] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[(NTHREADS*6)+threadIdx.x] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[(NTHREADS*7)+threadIdx.x] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);
    vertlist[(NTHREADS*8)+threadIdx.x] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[(NTHREADS*9)+threadIdx.x] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[(NTHREADS*10)+threadIdx.x] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[(NTHREADS*11)+threadIdx.x] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
    __syncthreads();
#else

    float3 vertlist[12];

    vertlist[0] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[1] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[2] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[3] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);

    vertlist[4] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[5] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[6] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[7] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);

    vertlist[8] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[9] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[10] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[11] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
#endif

    // output triangle vertices
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    for (int i=0; i<numVerts; i+=3)
    {
        uint index = numVertsScanned[voxel] + i;

        float3 *v[3];
        uint edge;
        edge = tex1Dfetch(triTex, (cubeindex*16) + i);
#if USE_SHARED
        v[0] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[0] = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 1);
#if USE_SHARED
        v[1] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[1] = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 2);
#if USE_SHARED
        v[2] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[2] = &vertlist[edge];
#endif

        // calculate triangle surface normal
        float3 n = calcNormal(v[0], v[1], v[2]);

        if (index < (maxVerts - 3))
        {
            pos[index] = make_float4(*v[0], 1.0f);
            norm[index] = make_float4(n, 0.0f);

            pos[index+1] = make_float4(*v[1], 1.0f);
            norm[index+1] = make_float4(n, 0.0f);

            pos[index+2] = make_float4(*v[2], 1.0f);
            norm[index+2] = make_float4(n, 0.0f);
        }
    }
}

extern "C" void
launch_generateTriangles(dim3 grid, dim3 threads,
                          float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, uchar *volume,
                          uint3 gridSize, float isoValue, uint activeVoxels, uint maxVerts)
{
    generateTriangles<<<grid, NTHREADS>>>(pos, norm,
                                           compactedVoxelArray,
                                           numVertsScanned, volume,
                                           gridSize, isoValue, activeVoxels,
                                           maxVerts);
    getLastCudaError("generateTriangles2 failed");
}

extern "C" void ThrustScanWrapper(unsigned int *output, unsigned int *input, unsigned int numElements)
{
    thrust::exclusive_scan(thrust::device_ptr<unsigned int>(input),
                           thrust::device_ptr<unsigned int>(input + numElements),
                           thrust::device_ptr<unsigned int>(output));
}

#endif
